#pragma once
#include <iostream>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include <ctime>


#include <thrust/system/cuda/execution_policy.h>

#include <iostream>

// DSIZE determines duration of H2D and D2H transfers
// #define DSIZE (1048576*8)
// // SSIZE,LSIZE determine duration of kernel launched by thrust
// #define SSIZE (1024*512)
// #define LSIZE 1
// // KSIZE determines size of thrust kernels (number of threads per block)
// #define KSIZE 64
// #define TV1 1
// #define TV2 2

// typedef int mytype;
// typedef thrust::host_vector<mytype, thrust::cuda::experimental::pinned_allocator<mytype> > pinnedVector;



// #include "cuMerge.h"
void test(double* data, int len) {

	//thrust::sort(data, data+len);
	thrust::sort(thrust::device_pointer_cast(data), thrust::device_pointer_cast(data+len));
}


int main() {
	int n = 1024;
	double* data = new double[n];
	double* data_d;
	hipEvent_t start, stop;
	//struct timespec start, stop;
	hipStream_t stream;
	hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((void**)&data_d, n*sizeof(double));
	for(int i = 0; i < n; ++i){
	
		data[i] = rand() % 10000;
	}
	hipEventRecord(start, 0);
	//hipMemcpy(data_d, data, n*sizeof(double), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	//clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
	hipMemcpy(data_d, data, n*sizeof(double), hipMemcpyHostToDevice);
	//thrust::sort(thrust::device_pointer_cast(data_d), thrust::device_pointer_cast(data_d + n));
	//hipEventSynchronize(stop);
	//thrust::sort(data, data+n);
	//test(data_d, n);
	thrust::sort(thrust::cuda::par.on(stream), thrust::device_pointer_cast(data_d), thrust::device_pointer_cast(data_d+n));
	// gsort(data_d, n);
	// hipMemcpy(data, data_d, n*sizeof(double), hipMemcpyDeviceToHost);
	//clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
	//hipMemcpy(data, data_d, n*sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float times;
	hipEventElapsedTime(&times, start, stop);
	//times = (stop.tv_sec-start.tv_sec)*1e3 + (stop.tv_nsec- start.tv_nsec)/1e6;
	std::cout << times << std::endl;
	// for(int i = 0; i < n; ++i){
	// 	std::cout << data[i] << " ";
	// }
	hipEventDestroy(stop);
	hipEventDestroy(start);
	hipStreamDestroy(stream);
	delete data;
	hipFree(data_d);
	return 0;

}
