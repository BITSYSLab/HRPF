#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <map>
#include <cstdlib>
#include <algorithm>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <string>
#include <time.h> //for srand
#include <limits.h>
#include "datastructture/matrix.h"
#include "datastructture/arraylist.h"
#include "algorithm/parallel_for_zero/parallel_for_inc.h"
#include "framework/framework.h"
#include "tool/helper.h"
// #include "algorithm/parallel_for/parallel_for_harness.hpp"
#include <omp.h>
#include <sys/time.h>
using namespace std;
#define cols 8
static int rows;
#define k 3

struct UserData_t : public Basedata_t
{
public:
    UserData_t(std::vector<ArrayList *> buf) : v_buffer(buf)
    {
    }

public:
    std::vector<Matrix *> m_buffer;
    std::vector<ArrayList *> v_buffer;
};

void cfor_func(Basedata_t *data)
{
    // std::cout << "cpu" << std::endl;
    auto d = (loopData_t *)data;
    auto a0 = ((UserData_t *)(d->buffer))->v_buffer[0]->get_cdata();
    auto a1 = ((UserData_t *)(d->buffer))->v_buffer[1]->get_cdata();
    double *cent[3];
    cent[0] = ((UserData_t *)(d->buffer))->v_buffer[2]->get_cdata();
    cent[1] = ((UserData_t *)(d->buffer))->v_buffer[3]->get_cdata();
    cent[2] = ((UserData_t *)(d->buffer))->v_buffer[4]->get_cdata();
    // cent[3] = ((UserData_t*)(d->buffer))->v_buffer[3]->get_cdata();
    auto dist = ((UserData_t *)(d->buffer))->v_buffer[5]->get_cdata();
    auto index = ((UserData_t *)(d->buffer))->v_buffer[6]->get_cdata();

    // size_t lda = ((UserData_t*)(d->buffer))->m_buffer[0]->get_ld();

    size_t s_i = d->start;
    size_t e_i = d->end;
    size_t s_j = 0;
    size_t e_j = cols;
// std::cout << s_i << s_j << e_i << e_j << std::endl;
#pragma omp parallel for num_threads(16)
    for (int i = s_i; i < e_i; ++i)
    {
        int minIdx = -1;
        double minDst = INT_MAX;

        for (int j = 0; j < 3; ++j)
        {
            double sum = 0;
            // for(int c = s_j; c < e_j; ++c){
            sum += sqrt((a0[i] - cent[j][0]) * (a0[i] - cent[j][0]));
            sum += sqrt((a1[i] - cent[j][1]) * (a1[i] - cent[j][1]));
            // }
            if (sum < minDst)
            {
                minDst = sum;
                minIdx = j;
            }
        }
        dist[i] = minDst;
        index[i] = minIdx;
    }
}

__global__ void kernel_2DKMS(size_t s_i, size_t e_i, size_t s_j, size_t e_j,
                             size_t lda, size_t ldb, size_t ldc,
                             size_t chunk, double *a0, double *a1, double *cent0, double *cent1,
                             double *cent2, double *dist, double *index)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double *cent[3] = {cent0, cent1, cent2};
    int start_i = s_i + tid * chunk;
    int end_i = start_i + chunk < e_i ? start_i + chunk : e_i;

    for (int i = start_i; i < end_i; ++i)
    {
        int minIdx = -1;
        double minDst = INT_MAX;

        for (int j = 0; j < 3; ++j)
        {
            double sum = 0;
            // for(int c = s_j; c < e_j; ++c){
            //     sum += sqrt((a[i + c*lda] - cent[j][c])*(a[i + c*lda] - cent[j][c]));
            // }
            sum += sqrt((a0[i] - cent[j][0]) * (a0[i] - cent[j][0]));
            sum += sqrt((a1[i] - cent[j][1]) * (a1[i] - cent[j][1]));
            if (sum < minDst)
            {
                minDst = sum;
                minIdx = j;
            }
        }

        dist[i] = minDst;
        index[i] = minIdx;
    }
}

void gfor_func(Basedata_t *data)
{
    // std::cout << "gpu" << std::endl;
    auto d = (loopData_t *)data;
    auto a0 = ((UserData_t *)(d->buffer))->v_buffer[0]->get_gdata();
    auto a1 = ((UserData_t *)(d->buffer))->v_buffer[1]->get_gdata();
    auto cent0 = ((UserData_t *)(d->buffer))->v_buffer[2]->get_gdata();
    auto cent1 = ((UserData_t *)(d->buffer))->v_buffer[3]->get_gdata();
    auto cent2 = ((UserData_t *)(d->buffer))->v_buffer[4]->get_gdata();
    // auto cent3 = ((UserData_t*)(d->buffer))->v_buffer[3]->get_gdata();
    auto dist = ((UserData_t *)(d->buffer))->v_buffer[5]->get_gdata();
    auto index = ((UserData_t *)(d->buffer))->v_buffer[6]->get_gdata();

    // size_t lda = ((UserData_t*)(d->buffer))->m_buffer[0]->get_ld();

    size_t s_i = d->start;
    size_t e_i = d->end;
    size_t s_j = 0;
    size_t e_j = cols;

    int blocks_required = 1;
    int threads_per_block = 1024;
    int chunk_size = 1;
    int size = e_i - s_i;
    if (size % (threads_per_block * chunk_size))
    {
        blocks_required = size / (threads_per_block * chunk_size) + 1;
    }
    else
    {
        blocks_required = size / (threads_per_block * chunk_size);
    }
    hipStream_t stream_ = stream();
    kernel_2DKMS<<<blocks_required, threads_per_block, 0, stream_>>>(s_i, e_i, s_j, e_j, 0, 0, 0,
                                                                     chunk_size, a0, a1, cent0, cent1, cent2, dist, index);
}

class KMEANS
{
private:
    double *dataSet[cols]; // rows*cols

    double *centroids[k]; // k*cols
    double *perMinDist;   // rows
    double *perIndex;     // rows
    typedef struct MinMax
    {
        double Min;
        double Max;
        MinMax(double min, double max) : Min(min), Max(max) {}
    } tMinMax;

    typedef struct Node
    {
        int minIndex;
        double minDist;
        Node(int idx, double dist) : minIndex(idx), minDist(dist) {}
    } tNode;
    std::vector<tNode> clusterAssment;

    tMinMax getMinMax(int idx);
    void setCentroids(tMinMax &tminmax, int idx);
    void initClusterAssment();

public:
    // double* train, double* index, double* dist, double* centr[k]
    KMEANS()
    {
        // dataSet = train;
        // perMinDist = dist;
        // perIndex = index;
        // centroids = centr;
        // for(int i = 0; i < k; ++i){
        //     centroids[i] = centr[i];
        // }
    }
    void loadDataSet();
    void kmeans();
    void randCent();
};

void KMEANS::initClusterAssment()
{
    tNode node(-1, -1);
    for (int i = 0; i < rows; ++i)
    {
        clusterAssment.push_back(node);
    }
}

void KMEANS::kmeans()
{
    initClusterAssment();
    // bool clusterChanged = true;
    int clusterChanged = 20;
    // Matrix* trainData = new Matrix(rows, cols);
    ArrayList *trainData0 = new ArrayList(rows);
    dataSet[0] = trainData0->get_cdata();
    ArrayList *trainData1 = new ArrayList(rows);
    dataSet[1] = trainData1->get_cdata();
    ArrayList *pminDist = new ArrayList(rows);
    perMinDist = pminDist->get_cdata();
    ArrayList *pminIndex = new ArrayList(rows);
    perIndex = pminIndex->get_cdata();
    ArrayList *cent[k];
    for (int i = 0; i < k; ++i)
    {
        cent[i] = new ArrayList(cols);
        centroids[i] = cent[i]->get_cdata();
    }
    loadDataSet();
    std::cout << "load dataset succ" << std::endl;
    randCent();
    UserData_t *user = new UserData_t({trainData0, trainData1, cent[0], cent[1], cent[2], pminDist, pminIndex});
    struct timeval start, end;
    gettimeofday(&start, NULL);
    while (clusterChanged)
    {
        clusterChanged--; //= false;

        Framework::init();

        parallelForI2D(new loopData_t(0, rows, user), cfor_func, gfor_func);
// pminDist->access(Runtime::get_instance().get_cpu(), MemAccess::R);
// pminIndex->access(Runtime::get_instance().get_cpu(), MemAccess::R);
#pragma omp parallel for num_threads(16)
        for (int i = 0; i < rows; ++i)
        {
            if (clusterAssment[i].minIndex != (int)perIndex[i])
            {
                // clusterChanged = true;
                clusterAssment[i].minIndex = (int)perIndex[i];
                clusterAssment[i].minDist = perMinDist[i];
            }
        }

        for (int c = 0; c < k; ++c)
        {
            std::vector<double> vec(cols, 0);
            int cnt = 0;
            for (int i = 0; i < rows; ++i)
            {
                if (clusterAssment[i].minIndex == c)
                {
                    ++cnt;
                    for (int j = 0; j < cols; ++j)
                    {
                        vec[j] += dataSet[j][i];
                    }
                }
            }

            for (int i = 0; i < cols; ++i)
            {
                if (cnt)
                    vec[i] /= cnt;
                centroids[c][i] = vec[i];
            }
        }
        //
    }
    gettimeofday(&end, NULL);
    double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
    std::cout << seconds << std::endl;
    // UserData_t* user = new UserData_t({trainData},{cent[0], cent[1], cent[2], cent[3], pminDist, pminIndex});
    std::ofstream ofs;            //定义流对象
    ofs.open("km.txt", ios::out); //以写的方式打开文件
    for (int i = 0; i < rows; ++i)
    {
        ofs << clusterAssment[i].minIndex << std::endl;
    }
    ofs.close();

    delete user;
    delete trainData0;
    delete trainData1;
    delete pminDist;
    delete pminIndex;
    for (int i = 0; i < k; ++i)
    {
        delete cent[i];
    }
}

void KMEANS::setCentroids(tMinMax &tminmax, int idx)
{
    double rangeIdx = tminmax.Max - tminmax.Min;
    for (int i = 0; i < k; ++i)
    {
        // std::cout << "k:" << i << std::endl;
        centroids[i][idx] = tminmax.Min + rangeIdx * (rand() / (double)RAND_MAX);
    }
}

typename KMEANS::tMinMax KMEANS::getMinMax(int idx)
{
    double min, max;
    dataSet[idx][0] > dataSet[idx][1] ? (max = dataSet[idx][0], min = dataSet[idx][1]) : (max = dataSet[idx][1], min = dataSet[idx][0]);

    for (int i = 2; i < rows; ++i)
    {
        if (dataSet[idx][i] < min)
            min = dataSet[idx][i];
        else if (dataSet[idx][i] < max)
            max = dataSet[idx][i];
        else
            continue;
    }

    tMinMax tminmax(min, max);
    // std::cout << "finish..." << min << max << std::endl;
    return tminmax;
}

void KMEANS::randCent()
{
    srand(time(NULL));
    for (int j = 0; j < cols; ++j)
    {
        // std::cout << "cols:" << j << std::endl;
        tMinMax tminmax = getMinMax(j);
        setCentroids(tminmax, j);
    }
}

void KMEANS::loadDataSet()
{
    std::ifstream fin;
    fin.open("kmdata.txt");

    if (!fin)
    {
        std::cout << "can not open the file data.txt" << std::endl;
        exit(1);
    }

    /* input the dataSet */
    for (int i = 0; i < rows; i++)
    {
        // for(int j=0;j<cols;j++)
        // {
        fin >> dataSet[0][i] >> dataSet[1][i];
        // }
    }
    fin.close();
}

int main(int argc, char **argv)
{
    rows = std::atoi(argv[1]);
    // double* train;
    // hipHostAlloc(&train, rows*cols * sizeof(double), hipHostMallocMapped);
    // double* pIndex;
    // hipHostAlloc(&pIndex, rows * sizeof(double), hipHostMallocMapped);
    // double* pMinDist;
    // hipHostAlloc(&pMinDist, rows * sizeof(double), hipHostMallocMapped);
    // double* centr[k];
    // for(int i = 0; i < k; ++i){
    //     hipHostAlloc(&centr[i], cols * sizeof(double), hipHostMallocMapped);;
    // }train, pIndex, pMinDist, centr
    KMEANS kms;
    // std::cout << "kms construct..." << std::endl;
    // kms.loadDataSet();
    // // std::cout << "load dataset succ" << std::endl;
    // kms.randCent();
    // std::cout << "rand centroid" << std::endl;
    kms.kmeans();
    return 0;
}
